
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    printf("Hello World From GPU in Block: (%d, %d), Thread: (%d, %d)!!\n", bix, biy, tix, tiy);
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!!\n");

    helloFromGPU<<< 2, 10 >>>();
    hipDeviceReset();

    return 0;
}
